#include "hip/hip_runtime.h"
#include <algorithm>
#include <queue>
#include <random>
#include <chrono>

#include <glm/gtx/hash.hpp>

#include "casdf.hpp"
#include "microlog.h"

// Bounding box of mesh
// TODO: rearrange into cpp and cuda files
static std::pair <glm::vec3, glm::vec3> bound(const geometry &g)
{
	glm::vec3 max = g.vertices[0];
	glm::vec3 min = g.vertices[0];
	for (const glm::vec3 &v : g.vertices) {
		max = glm::max(max, v);
		min = glm::min(min, v);
	}

	return { max, min };
}

// Closest point on triangle
__forceinline__ __host__ __device__
void triangle_closest_point(const glm::vec3 &v0, const glm::vec3 &v1, const glm::vec3 &v2, const glm::vec3 &p, glm::vec3 *closest, glm::vec3 *bary, float *distance)
{
	glm::vec3 B = v0;
	glm::vec3 E1 = v1 - v0;
	glm::vec3 E2 = v2 - v0;
	glm::vec3 D = B - p;

	float a = glm::dot(E1, E1);
	float b = glm::dot(E1, E2);
	float c = glm::dot(E2, E2);
	float d = glm::dot(E1, D);
	float e = glm::dot(E2, D);
	float f = glm::dot(D, D);

	float det = a * c - b * b;
	float s = b * e - c * d;
	float t = b * d - a * e;

	if (s + t <= det) {
		if (s < 0.0f) {
			if (t < 0.0f) {
				if (d < 0.0f) {
					s = glm::clamp(-d / a, 0.0f, 1.0f);
					t = 0.0f;
				} else {
					s = 0.0f;
					t = glm::clamp(-e / c, 0.0f, 1.0f);
				}
			} else {
				s = 0.0f;
				t = glm::clamp(-e / c, 0.0f, 1.0f);
			}
		} else if (t < 0.0f) {
			s = glm::clamp(-d / a, 0.0f, 1.0f);
			t = 0.0f;
		} else {
			float invDet = 1.0f / det;
			s *= invDet;
			t *= invDet;
		}
	} else {
		if (s < 0.0f) {
			float tmp0 = b + d;
			float tmp1 = c + e;
			if (tmp1 > tmp0) {
				float numer = tmp1 - tmp0;
				float denom = a - 2 * b + c;
				s = glm::clamp(numer / denom, 0.0f, 1.0f);
				t = 1 - s;
			} else {
				t = glm::clamp(-e / c, 0.0f, 1.0f);
				s = 0.0f;
			}
		} else if (t < 0.0f) {
			if (a + d > b + e) {
				float numer = c + e - b - d;
				float denom = a - 2 * b + c;
				s = glm::clamp(numer / denom, 0.0f, 1.0f);
				t = 1 - s;
			} else {
				s = glm::clamp(-e / c, 0.0f, 1.0f);
				t = 0.0f;
			}
		} else {
			float numer = c + e - b - d;
			float denom = a - 2 * b + c;
			s = glm::clamp(numer / denom, 0.0f, 1.0f);
			t = 1.0f - s;
		}
	}

	*closest = B + s * E1 + t * E2;
	*bary = glm::vec3(1.0f - s - t, s, t);
	*distance = glm::length(*closest - p);
}

__forceinline__ __host__ __device__
glm::uvec3 pcg(glm::uvec3 v)
{
	v = v * 1664525u + 1013904223u;
	v.x += v.y * v.z;
	v.y += v.z * v.x;
	v.z += v.x * v.y;
	v ^= v >> 16u;
	v.x += v.y * v.z;
	v.y += v.z * v.x;
	v.z += v.x * v.y;
	return v;
}

__forceinline__ __host__ __device__
glm::vec3 pcg(glm::vec3 v)
{
	glm::uvec3 u = *(glm::uvec3 *) &v;
	u = pcg(u);
	u &= glm::uvec3(0x007fffffu);
	u |= glm::uvec3(0x3f800000u);
	return *(glm::vec3 *) &u;
}

__forceinline__ __host__ __device__
float bbox_distance(glm::vec3 p, glm::vec3 min, glm::vec3 max)
{
	glm::vec3 dmin = glm::abs(p - min);
	glm::vec3 dmax = glm::abs(p - max);
	glm::vec3 d = glm::min(dmin, dmax);
	float mind = glm::min(d.x, glm::min(d.y, d.z));

	bool inside = (p.x >= min.x && p.x <= max.x)
		&& (p.y >= min.y && p.y <= max.y)
		&& (p.z >= min.z && p.z <= max.z);

	return inside ? 0.0f : mind;
}

// GPU kernels
__global__
static void brute_closest_point_kernel(cumesh cu_mesh, closest_point_kinfo kinfo)
{
	uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
	uint32_t stride = blockDim.x * gridDim.x;

	for (uint32_t i = tid; i < kinfo.point_count; i += stride) {
		glm::vec3 point = kinfo.points[i];
		glm::vec3 closest;
		glm::vec3 barycentrics;
		uint32_t triangle;

		float min_distance = FLT_MAX;
		for (uint32_t j = 0; j < cu_mesh.triangle_count; j++) {
			glm::uvec3 tri = cu_mesh.triangles[j];

			glm::vec3 v0 = cu_mesh.vertices[tri.x];
			glm::vec3 v1 = cu_mesh.vertices[tri.y];
			glm::vec3 v2 = cu_mesh.vertices[tri.z];

			// Rough culling
			// glm::vec3 min = glm::min(v0, glm::min(v1, v2));
			// glm::vec3 max = glm::max(v0, glm::max(v1, v2));
			//
			// if (bbox_distance(point, min, max) > min_distance)
			// 	continue;

			glm::vec3 candidate;
			glm::vec3 bary;
			float distance;

			triangle_closest_point(v0, v1, v2, point, &candidate, &bary, &distance);

			if (distance < min_distance) {
				min_distance = distance;
				closest = candidate;
				barycentrics = bary;
				triangle = j;
			}
		}

		kinfo.closest[i] = closest;
		kinfo.bary[i] = barycentrics;
		kinfo.triangles[i] = triangle;
	}
}

__global__
static void closest_point_kernel(dev_cas_grid cas, closest_point_kinfo kinfo)
{
	uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
	uint32_t stride = blockDim.x * gridDim.x;

	for (uint32_t i = tid; i < kinfo.point_count; i += stride) {
		glm::vec3 point = kinfo.points[i];
		glm::vec3 closest;
		uint32_t triangle;

		glm::vec3 bin_flt = glm::clamp((point - cas.min) / cas.bin_size,
				glm::vec3(0), glm::vec3(cas.resolution - 1));

		glm::ivec3 bin = glm::ivec3(bin_flt);
		uint32_t bin_index = bin.x + bin.y * cas.resolution + bin.z * cas.resolution * cas.resolution;

		uint32_t index0 = cas.index0[bin_index];
		uint32_t index1 = cas.index1[bin_index];

		float min_distance = FLT_MAX;
		for (uint32_t j = index0; j < index1; j++) {
			uint32_t triangle_index = cas.query_triangles[j];
			glm::uvec3 tri = cas.triangles[triangle_index];

			glm::vec3 v0 = cas.vertices[tri.x];
			glm::vec3 v1 = cas.vertices[tri.y];
			glm::vec3 v2 = cas.vertices[tri.z];

			// TODO: prune triangles that are too far away (based on bbox)?
			glm::vec3 candidate;
			glm::vec3 bary;
			float distance;

			triangle_closest_point(v0, v1, v2, point, &candidate, &bary, &distance);

			if (distance < min_distance) {
				min_distance = distance;
				closest = candidate;
				triangle = triangle_index;
			}
		}

		kinfo.closest[i] = closest;
		kinfo.triangles[i] = triangle;
	}
}

__global__
void sample_kernel(sample_result result, cumesh mesh, float time)
{
	uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
	uint32_t stride = blockDim.x * gridDim.x;

	for (uint32_t i = tid; i < result.point_count; i += stride) {
		glm::uvec3 seed0 = mesh.triangles[i % mesh.triangle_count];
		glm::vec3 seed1 = mesh.vertices[i % mesh.vertex_count];

		uint32_t tint = *(uint32_t *) &time;
		seed0.x ^= tint;
		seed0.y ^= tint;
		seed0.z ^= tint;

		seed1.x *= __sinf(time);
		seed1.y *= __sinf(time);
		seed1.z *= __sinf(time);

		glm::uvec3 tri = pcg(seed0);
		glm::vec3 bary = pcg(seed1);

		uint32_t tindex = tri.x % mesh.triangle_count;
		tri = mesh.triangles[tindex];

		glm::vec3 v0 = mesh.vertices[tri.x];
		glm::vec3 v1 = mesh.vertices[tri.y];
		glm::vec3 v2 = mesh.vertices[tri.z];

		bary = glm::normalize(bary);
		bary.x = 1.0f - bary.y - bary.z;

		result.points[i] = bary.x * v0 + bary.y * v1 + bary.z * v2;
		result.barys[i] = bary;
		result.indices[i] = tindex;
	}
}

// Allocate cumeshes
cumesh cumesh_alloc(const geometry &mesh)
{
	cumesh cu_mesh;
	cu_mesh.vertex_count = mesh.vertices.size();
	cu_mesh.triangle_count = mesh.triangles.size();

	hipMalloc(&cu_mesh.vertices, sizeof(glm::vec3) * cu_mesh.vertex_count);
	hipMalloc(&cu_mesh.triangles, sizeof(glm::uvec3) * cu_mesh.triangle_count);

	hipMemcpy(cu_mesh.vertices, mesh.vertices.data(),
		sizeof(glm::vec3) * cu_mesh.vertex_count, hipMemcpyHostToDevice);

	hipMemcpy(cu_mesh.triangles, mesh.triangles.data(),
		sizeof(glm::uvec3) * cu_mesh.triangle_count, hipMemcpyHostToDevice);

	return cu_mesh;
}

void cumesh_reload(cumesh cu_mesh, const geometry &mesh)
{
	if (cu_mesh.vertex_count != mesh.vertices.size()) {
		hipFree(cu_mesh.vertices);
		hipMalloc(&cu_mesh.vertices, sizeof(glm::vec3) * mesh.vertices.size());
		cu_mesh.vertex_count = mesh.vertices.size();
	}

	if (cu_mesh.triangle_count != mesh.triangles.size()) {
		hipFree(cu_mesh.triangles);
		hipMalloc(&cu_mesh.triangles, sizeof(glm::uvec3) * mesh.triangles.size());
		cu_mesh.triangle_count = mesh.triangles.size();
	}

	hipMemcpy(cu_mesh.vertices, mesh.vertices.data(),
		sizeof(glm::vec3) * mesh.vertices.size(), hipMemcpyHostToDevice);

	hipMemcpy(cu_mesh.triangles, mesh.triangles.data(),
		sizeof(glm::uvec3) * mesh.triangles.size(), hipMemcpyHostToDevice);
}

// Allocating sample information
sample_result sample_result_alloc(uint32_t point_count, compute_api mode)
{
	sample_result result;
	result.point_count = point_count;
	result.api = mode;

	if (mode == eCPU) {
		result.points = new glm::vec3[point_count];
		result.barys = new glm::vec3[point_count];
		result.indices = new uint32_t[point_count];
	} else if (mode == eCUDA) {
		hipMalloc(&result.points, sizeof(glm::vec3) * point_count);
		hipMalloc(&result.barys, sizeof(glm::vec3) * point_count);
		hipMalloc(&result.indices, sizeof(uint32_t) * point_count);
	}

	return result;
}

void sample(sample_result result, const geometry &g, float time)
{
	ULOG_ASSERT(result.api == eCPU);
	for (uint32_t i = 0; i < result.point_count; i++) {
		glm::uvec3 seed0 = g.triangles[i % g.triangles.size()];
		glm::vec3 seed1 = g.vertices[i % g.vertices.size()];

		uint32_t tint = *(uint32_t *) &time;
		seed0.x ^= tint;
		seed0.y ^= tint;
		seed0.z ^= tint;

		seed1.x *= sinf(time);
		seed1.y *= sinf(time);
		seed1.z *= sinf(time);

		glm::uvec3 tri = pcg(seed0);
		glm::vec3 bary = pcg(seed1);

		uint32_t tindex = tri.x % g.triangles.size();
		tri = g.triangles[tindex];

		glm::vec3 v0 = g.vertices[tri.x];
		glm::vec3 v1 = g.vertices[tri.y];
		glm::vec3 v2 = g.vertices[tri.z];

		bary = glm::normalize(bary);
		bary.x = 1.0f - bary.y - bary.z;

		result.points[i] = bary.x * v0 + bary.y * v1 + bary.z * v2;
		result.barys[i] = bary;
		result.indices[i] = tindex;
	}
}

void sample(sample_result result, const cumesh &mesh, float time)
{
	ULOG_ASSERT(result.api == eCUDA);

	dim3 block(256);
	dim3 grid((result.point_count + block.x - 1) / block.x);
	sample_kernel <<< grid, block >>> (result, mesh, time);
}

void memcpy(sample_result dst, const sample_result &src)
{
	if (dst.api == eCPU && src.api == eCPU) {
		memcpy(dst.points, src.points, sizeof(glm::vec3) * dst.point_count);
		memcpy(dst.barys, src.barys, sizeof(glm::vec3) * dst.point_count);
		memcpy(dst.indices, src.indices, sizeof(uint32_t) * dst.point_count);
	} else if (dst.api == eCUDA && src.api == eCUDA) {
		hipMemcpy(dst.points, src.points, sizeof(glm::vec3) * dst.point_count, hipMemcpyDeviceToDevice);
		hipMemcpy(dst.barys, src.barys, sizeof(glm::vec3) * dst.point_count, hipMemcpyDeviceToDevice);
		hipMemcpy(dst.indices, src.indices, sizeof(uint32_t) * dst.point_count, hipMemcpyDeviceToDevice);
	} else if (dst.api == eCPU && src.api == eCUDA) {
		hipMemcpy(dst.points, src.points, sizeof(glm::vec3) * dst.point_count, hipMemcpyDeviceToHost);
		hipMemcpy(dst.barys, src.barys, sizeof(glm::vec3) * dst.point_count, hipMemcpyDeviceToHost);
		hipMemcpy(dst.indices, src.indices, sizeof(uint32_t) * dst.point_count, hipMemcpyDeviceToHost);
	} else if (dst.api == eCUDA && src.api == eCPU) {
		hipMemcpy(dst.points, src.points, sizeof(glm::vec3) * dst.point_count, hipMemcpyHostToDevice);
		hipMemcpy(dst.barys, src.barys, sizeof(glm::vec3) * dst.point_count, hipMemcpyHostToDevice);
		hipMemcpy(dst.indices, src.indices, sizeof(uint32_t) * dst.point_count, hipMemcpyHostToDevice);
	}
}

// Allocate kinfo
closest_point_kinfo closest_point_kinfo_alloc(uint32_t point_count, compute_api mode)
{
	closest_point_kinfo kinfo;
	kinfo.point_count = point_count;
	kinfo.api = mode;

	if (mode == eCPU) {
		kinfo.points = new glm::vec3[point_count];
		kinfo.closest = new glm::vec3[point_count];
		kinfo.bary = new glm::vec3[point_count];
		kinfo.distances = new float[point_count];
		kinfo.triangles = new uint32_t[point_count];
	} else if (mode == eCUDA) {
		hipMalloc(&kinfo.points, point_count * sizeof(glm::vec3));
		hipMalloc(&kinfo.closest, point_count * sizeof(glm::vec3));
		hipMalloc(&kinfo.bary, point_count * sizeof(glm::vec3));
		hipMalloc(&kinfo.distances, point_count * sizeof(float));
		hipMalloc(&kinfo.triangles, point_count * sizeof(uint32_t));
	}

	return kinfo;
}

void memcpy(closest_point_kinfo dst, const closest_point_kinfo &src)
{
	if (dst.api == eCPU && src.api == eCPU) {
		memcpy(dst.points, src.points, sizeof(glm::vec3) * dst.point_count);
		memcpy(dst.closest, src.closest, sizeof(glm::vec3) * dst.point_count);
		memcpy(dst.bary, src.bary, sizeof(glm::vec3) * dst.point_count);
		memcpy(dst.distances, src.distances, sizeof(float) * dst.point_count);
		memcpy(dst.triangles, src.triangles, sizeof(uint32_t) * dst.point_count);
	} else if (dst.api == eCUDA && src.api == eCUDA) {
		hipMemcpy(dst.points, src.points, sizeof(glm::vec3) * dst.point_count, hipMemcpyDeviceToDevice);
		hipMemcpy(dst.closest, src.closest, sizeof(glm::vec3) * dst.point_count, hipMemcpyDeviceToDevice);
		hipMemcpy(dst.bary, src.bary, sizeof(glm::vec3) * dst.point_count, hipMemcpyDeviceToDevice);
		hipMemcpy(dst.distances, src.distances, sizeof(float) * dst.point_count, hipMemcpyDeviceToDevice);
		hipMemcpy(dst.triangles, src.triangles, sizeof(uint32_t) * dst.point_count, hipMemcpyDeviceToDevice);
	} else if (dst.api == eCPU && src.api == eCUDA) {
		hipMemcpy(dst.points, src.points, sizeof(glm::vec3) * dst.point_count, hipMemcpyDeviceToHost);
		hipMemcpy(dst.closest, src.closest, sizeof(glm::vec3) * dst.point_count, hipMemcpyDeviceToHost);
		hipMemcpy(dst.bary, src.bary, sizeof(glm::vec3) * dst.point_count, hipMemcpyDeviceToHost);
		hipMemcpy(dst.distances, src.distances, sizeof(float) * dst.point_count, hipMemcpyDeviceToHost);
		hipMemcpy(dst.triangles, src.triangles, sizeof(uint32_t) * dst.point_count, hipMemcpyDeviceToHost);
	} else if (dst.api == eCUDA && src.api == eCPU) {
		hipMemcpy(dst.points, src.points, sizeof(glm::vec3) * dst.point_count, hipMemcpyHostToDevice);
		hipMemcpy(dst.closest, src.closest, sizeof(glm::vec3) * dst.point_count, hipMemcpyHostToDevice);
		hipMemcpy(dst.bary, src.bary, sizeof(glm::vec3) * dst.point_count, hipMemcpyHostToDevice);
		hipMemcpy(dst.distances, src.distances, sizeof(float) * dst.point_count, hipMemcpyHostToDevice);
		hipMemcpy(dst.triangles, src.triangles, sizeof(uint32_t) * dst.point_count, hipMemcpyHostToDevice);
	}
}

// Brute force closest point
void brute_closest_point(const geometry &g, const closest_point_kinfo &kinfo)
{
	ULOG_ASSERT(kinfo.api == eCPU);

	#pragma omp parallel for
	for (uint32_t i = 0; i < kinfo.point_count; i++) {
		glm::vec3 point = kinfo.points[i];
		glm::vec3 closest;
		glm::vec3 barycentrics;
		uint32_t triangle;

		float min_distance = FLT_MAX;
		for (uint32_t j = 0; j < g.triangles.size(); j++) {
			glm::uvec3 tri = g.triangles[j];

			glm::vec3 v0 = g.vertices[tri.x];
			glm::vec3 v1 = g.vertices[tri.y];
			glm::vec3 v2 = g.vertices[tri.z];

			glm::vec3 candidate;
			glm::vec3 bary;
			float distance;

			triangle_closest_point(v0, v1, v2, point, &candidate, &bary, &distance);

			if (distance < min_distance) {
				min_distance = distance;
				closest = candidate;
				barycentrics = bary;
				triangle = j;
			}
		}

		kinfo.closest[i] = closest;
		kinfo.bary[i] = barycentrics;
		kinfo.distances[i] = min_distance;
		kinfo.triangles[i] = triangle;
	}
}

void brute_closest_point(const cumesh &cu_mesh, const closest_point_kinfo &kinfo)
{
	ULOG_ASSERT(kinfo.api == eCUDA);

	dim3 block(256);
	dim3 grid((kinfo.point_count + block.x - 1) / block.x);
	brute_closest_point_kernel <<< grid, block >>> (cu_mesh, kinfo);
}

// Cached acceleration structure
cas_grid::cas_grid(const geometry &ref_, uint32_t resolution_)
		: ref(ref_), resolution(resolution_)
{
	uint32_t size = resolution * resolution * resolution;
	overlapping_triangles.resize(size);
	query_triangles.resize(size);

	// Put triangles into bins
	std::tie(max, min) = bound(ref);
	glm::vec3 extent = { max.x - min.x, max.y - min.y, max.z - min.z };
	bin_size = extent / (float) resolution;

	for (size_t i = 0; i < ref.triangles.size(); i++) {
		const glm::uvec3 &triangle = ref.triangles[i];

		// Triangle belongs to all bins it intersects
		glm::vec3 v0 = ref.vertices[triangle.x];
		glm::vec3 v1 = ref.vertices[triangle.y];
		glm::vec3 v2 = ref.vertices[triangle.z];

		glm::vec3 tri_min = glm::min(glm::min(v0, v1), v2);
		glm::vec3 tri_max = glm::max(glm::max(v0, v1), v2);

		glm::vec3 min_bin = glm::clamp((tri_min - min) / bin_size, glm::vec3(0), glm::vec3(resolution - 1));
		glm::vec3 max_bin = glm::clamp((tri_max - min) / bin_size, glm::vec3(0), glm::vec3(resolution - 1));

		for (int x = min_bin.x; x <= max_bin.x; x++) {
			for (int y = min_bin.y; y <= max_bin.y; y++) {
				for (int z = min_bin.z; z <= max_bin.z; z++) {
					int index = x + y * resolution + z * resolution * resolution;
					overlapping_triangles[index].push_back(i);
				}
			}
		}
	}
}

uint32_t cas_grid::to_index(const glm::ivec3 &bin) const
{
	return bin.x + bin.y * resolution + bin.z * resolution * resolution;
}

uint32_t cas_grid::to_index(const glm::vec3 &p) const
{
	glm::vec3 bin_flt = glm::clamp((p - min) / bin_size, glm::vec3(0), glm::vec3(resolution - 1));
	glm::ivec3 bin = glm::ivec3(bin_flt);
	return to_index(bin);
}

// Find the complete set of query triangles for a point
std::unordered_set <uint32_t> cas_grid::closest_triangles(const glm::vec3 &p) const
{
	// Get the current bin
	glm::vec3 bin_flt = glm::clamp((p - min) / bin_size, glm::vec3(0), glm::vec3(resolution - 1));
	glm::ivec3 bin = glm::ivec3(bin_flt);
	uint32_t bin_index = to_index(p);

	// Find the closest non-empty bins
	std::vector <glm::ivec3> closest_bins;

	if (!overlapping_triangles[bin_index].empty()) {
		closest_bins.push_back(bin);
	} else {
		std::vector <glm::ivec3> plausible_bins;
		std::queue <glm::ivec3> queue;

		std::unordered_set <glm::ivec3> visited;
		bool stop = false;

		queue.push(bin);
		while (!queue.empty()) {
			glm::ivec3 current = queue.front();
			queue.pop();

			// If visited, continue
			if (visited.find(current) != visited.end())
				continue;

			visited.insert(current);

			// If non-empty, add to plausible bins and continue
			uint32_t current_index = current.x + current.y * resolution + current.z * resolution * resolution;
			if (!overlapping_triangles[current_index].empty()) {
				plausible_bins.push_back(current);

				// Also set the stop flag to stop adding neighbors
				stop = true;
				continue;
			}

			if (stop)
				continue;

			int dx[] = { -1, 0, 0, 1, 0, 0 };
			int dy[] = { 0, -1, 0, 0, 1, 0 };
			int dz[] = { 0, 0, -1, 0, 0, 1 };

			// Add all neighbors to queue...
			for (int i = 0; i < 6; i++) {
				glm::ivec3 next = current + glm::ivec3(dx[i], dy[i], dz[i]);
				if (next.x < 0 || next.x >= resolution ||
					next.y < 0 || next.y >= resolution ||
					next.z < 0 || next.z >= resolution)
					continue;

				// ...if not visited
				if (visited.find(next) == visited.end())
					queue.push(next);
			}
		}

		// Sort plausible bins by distance
		std::sort(plausible_bins.begin(), plausible_bins.end(),
			[&](const glm::ivec3 &a, const glm::ivec3 &b) {
				return glm::distance(bin_flt, glm::vec3(a)) < glm::distance(bin_flt, glm::vec3(b));
			}
		);

		assert(!plausible_bins.empty());

		// Add first one always; stop adding when difference is larger than voxel size
		closest_bins.push_back(plausible_bins[0]);
		for (uint32_t i = 1; i < plausible_bins.size(); i++) {
			glm::vec3 a = glm::vec3(plausible_bins[i - 1]);
			glm::vec3 b = glm::vec3(plausible_bins[i]);

			if (glm::distance(a, b) > 1.1f)
				break;

			closest_bins.push_back(plausible_bins[i]);
		}
	}

	assert(!closest_bins.empty());

	// Within the final collection, make sure to search immediate neighbors
	std::unordered_set <uint32_t> final_bins;

	for (const glm::ivec3 &bin : closest_bins) {
		int dx[] = { 0, -1, 0, 0, 1, 0, 0 };
		int dy[] = { 0, 0, -1, 0, 0, 1, 0 };
		int dz[] = { 0, 0, 0, -1, 0, 0, 1 };

		for (int i = 0; i < 7; i++) {
			glm::ivec3 next = bin + glm::ivec3(dx[i], dy[i], dz[i]);
			if (next.x < 0 || next.x >= resolution ||
				next.y < 0 || next.y >= resolution ||
				next.z < 0 || next.z >= resolution)
				continue;

			uint32_t next_index = to_index(next);
			if (!overlapping_triangles[next_index].empty())
				final_bins.insert(next_index);
		}
	}

	std::unordered_set <uint32_t> final_triangles;
	for (uint32_t bin_index : final_bins) {
		for (uint32_t index : overlapping_triangles[bin_index])
			final_triangles.insert(index);
	}

	return final_triangles;
}

// Load the cached query triangles if not already loaded
bool cas_grid::precache_query(const glm::vec3 &p)
{
	// Check if the bin is already cached
	uint32_t bin_index = to_index(p);
	// printf("  Precaching bin %d\n", bin_index);
	// printf("  p = (%f, %f, %f)\n", p.x, p.y, p.z);
	// printf("  max = (%f, %f, %f)\n", max.x, max.y, max.z);
	// printf("  min = (%f, %f, %f)\n", min.x, min.y, min.z);
	// printf("  bin size = (%f, %f, %f)\n", bin_size.x, bin_size.y, bin_size.z);
	ULOG_ASSERT(bin_index < query_triangles.size());

	if (!query_triangles[bin_index].empty())
		return false;

	// Otherwise, load the bin
	auto set = closest_triangles(p);
	query_triangles[bin_index] = query_bin(set.begin(), set.end());
	return true;
}

// Precache a collection of query points
float cas_grid::precache_query(const std::vector <glm::vec3> &points)
{
	uint32_t any_count = 0;
	for (const glm::vec3 &p : points)
		any_count += precache_query(p);

	return (float) any_count / (float) points.size();
}

// Single point query
std::tuple <glm::vec3, glm::vec3, float, uint32_t> cas_grid::query(const glm::vec3 &p) const
{
	// Assuming the point is precached already
	uint32_t bin_index = to_index(p);
	assert(bin_index < overlapping_triangles.size());

	const std::vector <uint32_t> &bin = query_triangles[bin_index];
	assert(bin.size() > 0);

	glm::vec3 closest = p;
	glm::vec3 barycentric;
	float distance = FLT_MAX;
	uint32_t triangle_index = 0;

	for (uint32_t index : bin) {
		const glm::uvec3 &tri = ref.triangles[index];
		glm::vec3 a = ref.vertices[tri[0]];
		glm::vec3 b = ref.vertices[tri[1]];
		glm::vec3 c = ref.vertices[tri[2]];

		glm::vec3 point;
		glm::vec3 bary;
		float dist;
		triangle_closest_point(a, b, c, p, &point, &bary, &dist);

		if (dist < distance) {
			closest = point;
			barycentric = bary;
			distance = dist;
			triangle_index = index;
		}
	}

	return std::make_tuple(closest, barycentric, distance, triangle_index);
}

// Host-side query
void cas_grid::query(const std::vector <glm::vec3> &sources,
		std::vector <glm::vec3> &closest,
		std::vector <glm::vec3> &bary,
		std::vector <float> &distance,
		std::vector <uint32_t> &triangle_index) const
{
	// Assuming all elements are precached already
	// and that the dst vector is already allocated
	ULOG_ASSERT(sources.size() == closest.size());
	ULOG_ASSERT(sources.size() == bary.size());
	ULOG_ASSERT(sources.size() == distance.size());
	ULOG_ASSERT(sources.size() == triangle_index.size());

	#pragma omp parallel for
	for (uint32_t i = 0; i < sources.size(); i++) {
		uint32_t bin_index = to_index(sources[i]);
		auto [c, b, d, t] = query(sources[i]);

		closest[i] = c;
		bary[i] = b;
		distance[i] = d;
		triangle_index[i] = t;
	}
}

void cas_grid::precache_device()
{
	dev_cas.min = min;
	dev_cas.max = max;
	dev_cas.bin_size = bin_size;

	dev_cas.resolution = resolution;
	dev_cas.vertex_count = ref.vertices.size();
	dev_cas.triangle_count = ref.triangles.size();

	std::vector <uint32_t> linear_query_triangles;
	std::vector <uint32_t> index0;
	std::vector <uint32_t> index1;

	uint32_t size = resolution * resolution * resolution;
	uint32_t offset = 0;

	for (uint32_t i = 0; i < size; i++) {
		uint32_t query_size = query_triangles[i].size();
		linear_query_triangles.insert(linear_query_triangles.end(),
				query_triangles[i].begin(),
				query_triangles[i].end());

		index0.push_back(offset);
		index1.push_back(offset + query_size);
		offset += query_size;
	}

	// Free old memory
	if (dev_cas.vertices != nullptr)
		hipFree(dev_cas.vertices);

	if (dev_cas.triangles != nullptr)
		hipFree(dev_cas.triangles);

	if (dev_cas.query_triangles != nullptr)
		hipFree(dev_cas.query_triangles);

	if (dev_cas.index0 != nullptr)
		hipFree(dev_cas.index0);

	if (dev_cas.index1 != nullptr)
		hipFree(dev_cas.index1);

	// Allocate new memory
	hipMalloc(&dev_cas.vertices, sizeof(glm::vec3) * ref.vertices.size());
	hipMalloc(&dev_cas.triangles, sizeof(glm::uvec3) * ref.triangles.size());

	hipMalloc(&dev_cas.query_triangles, sizeof(uint32_t) * linear_query_triangles.size());
	hipMalloc(&dev_cas.index0, sizeof(uint32_t) * index0.size());
	hipMalloc(&dev_cas.index1, sizeof(uint32_t) * index1.size());

	hipMemcpy(dev_cas.vertices, ref.vertices.data(), sizeof(glm::vec3) * ref.vertices.size(), hipMemcpyHostToDevice);
	hipMemcpy(dev_cas.triangles, ref.triangles.data(), sizeof(glm::uvec3) * ref.triangles.size(), hipMemcpyHostToDevice);

	hipMemcpy(dev_cas.query_triangles, linear_query_triangles.data(), sizeof(uint32_t) * linear_query_triangles.size(), hipMemcpyHostToDevice);
	hipMemcpy(dev_cas.index0, index0.data(), sizeof(uint32_t) * index0.size(), hipMemcpyHostToDevice);
	hipMemcpy(dev_cas.index1, index1.data(), sizeof(uint32_t) * index1.size(), hipMemcpyHostToDevice);
}

void cas_grid::query_device(closest_point_kinfo kinfo)
{
	dim3 block(256);
	dim3 grid((kinfo.point_count + block.x - 1) / block.x);

	closest_point_kernel <<< grid, block >>> (dev_cas, kinfo);
}

// Interior point query acceleration structure
__forceinline__ __host__ __device__
bool ray_x_triangle(glm::vec3 v0, glm::vec3 v1, glm::vec3 v2, glm::vec3 x, glm::vec3 d)
{
	glm::vec3 e1 = v1 - v0;
	glm::vec3 e2 = v2 - v0;
	glm::vec3 p = cross(d, e2);

	float a = dot(e1, p);
	if (std::abs(a) < 1e-6)
		return false;

	float f = 1.0 / a;
	glm::vec3 s = x - v0;
	float u = f * dot(s, p);

	if (u < 0.0 || u > 1.0)
		return false;

	glm::vec3 q = cross(s, e1);
	float v = f * dot(d, q);

	if (v < 0.0 || u + v > 1.0)
		return false;

	float t = f * dot(e2, q);
	return t > 1e-6;
}

float ipqas::query(const glm::vec3 &v) const
{
	static constexpr glm::vec3 dx { 1, 0, 0 };
	static constexpr glm::vec3 dy { 0, 1, 0 };
	static constexpr glm::vec3 dz { 0, 0, 1 };

	glm::vec3 normed = (v - ext_min) / (ext_max - ext_min);
	glm::vec3 voxel = normed * glm::vec3(resolution);
	voxel = glm::clamp(voxel, glm::vec3(0), glm::vec3(resolution - 1));

	uint32_t x = voxel.x;
	uint32_t y = voxel.y;
	uint32_t z = voxel.z;

	uint32_t xy_count = 0;
	uint32_t xy_count_neg = 0;

	for (uint32_t tindex : bins_xy[x + y * resolution]) {
		const glm::uvec3 &t = ref.triangles[tindex];
		const glm::vec3 &v0 = ref.vertices[t.x];
		const glm::vec3 &v1 = ref.vertices[t.y];
		const glm::vec3 &v2 = ref.vertices[t.z];

		if (ray_x_triangle(v0, v1, v2, v, dz))
			xy_count++;
	}

	if (xy_count % 2 == 0)
		return 1.0f;

	for (uint32_t tindex : bins_xy[x + y * resolution]) {
		const glm::uvec3 &t = ref.triangles[tindex];
		const glm::vec3 &v0 = ref.vertices[t.x];
		const glm::vec3 &v1 = ref.vertices[t.y];
		const glm::vec3 &v2 = ref.vertices[t.z];

		if (ray_x_triangle(v0, v1, v2, v, -dz))
			xy_count_neg++;
	}

	if (xy_count_neg % 2 == 0)
		return 1.0f;

	uint32_t xz_count = 0;
	uint32_t xz_count_neg = 0;

	for (uint32_t tindex : bins_xz[x + z * resolution]) {
		const glm::uvec3 &t = ref.triangles[tindex];
		const glm::vec3 &v0 = ref.vertices[t.x];
		const glm::vec3 &v1 = ref.vertices[t.y];
		const glm::vec3 &v2 = ref.vertices[t.z];

		if (ray_x_triangle(v0, v1, v2, v, dy))
			xz_count++;
	}

	if (xz_count % 2 == 0)
		return 1.0f;

	for (uint32_t tindex : bins_xz[x + z * resolution]) {
		const glm::uvec3 &t = ref.triangles[tindex];
		const glm::vec3 &v0 = ref.vertices[t.x];
		const glm::vec3 &v1 = ref.vertices[t.y];
		const glm::vec3 &v2 = ref.vertices[t.z];

		if (ray_x_triangle(v0, v1, v2, v, -dy))
			xz_count_neg++;
	}

	if (xz_count_neg % 2 == 0)
		return 1.0f;

	uint32_t yz_count = 0;
	uint32_t yz_count_neg = 0;

	for (uint32_t tindex : bins_yz[y + z * resolution]) {
		const glm::uvec3 &t = ref.triangles[tindex];
		const glm::vec3 &v0 = ref.vertices[t.x];
		const glm::vec3 &v1 = ref.vertices[t.y];
		const glm::vec3 &v2 = ref.vertices[t.z];

		if (ray_x_triangle(v0, v1, v2, v, dx))
			yz_count++;
	}

	if (yz_count % 2 == 0)
		return 1.0f;

	for (uint32_t tindex : bins_yz[y + z * resolution]) {
		const glm::uvec3 &t = ref.triangles[tindex];
		const glm::vec3 &v0 = ref.vertices[t.x];
		const glm::vec3 &v1 = ref.vertices[t.y];
		const glm::vec3 &v2 = ref.vertices[t.z];

		if (ray_x_triangle(v0, v1, v2, v, -dx))
			yz_count_neg++;
	}

	if (yz_count_neg % 2 == 0)
		return 1.0f;

	return -1.0f;
}

// Wholistic SDF routine
void sdf(const geometry &geometry, ipqas &interior_qas, closest_point_kinfo &kinfo, std::vector <float> &sdfs)
{
	ULOG_ASSERT(sdfs.size() == kinfo.point_count);
	brute_closest_point(geometry, kinfo);

	memcpy(sdfs.data(), kinfo.distances, kinfo.point_count * sizeof(float));

	std::vector <glm::vec3> points(kinfo.point_count);
	memcpy(points.data(), kinfo.points, kinfo.point_count * sizeof(glm::vec3));

	interior_qas.query(points, sdfs);
}

void sdf(cas_grid &cas, ipqas &interior_qas, const std::vector <glm::vec3> &points,
		std::vector <glm::vec3> &closest,
		std::vector <glm::vec3> &bary,
		std::vector <float> &sdfs,
		std::vector <uint32_t> &triangles)
{
	// TODO: size check
	cas.precache_query(points);
	cas.query(points, closest, bary, sdfs, triangles);
	interior_qas.query(points, sdfs);
}
