#include "hip/hip_runtime.h"
#include "common.hpp"

__global__
void remapper_kernel(const int32_t *__restrict__ map, glm::ivec3 *__restrict__ triangles, size_t size)
{
	size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
	size_t stride = blockDim.x * gridDim.x;

	for (size_t i = tid; i < size; i += stride) {
		triangles[i].x = map[triangles[i].x];
		triangles[i].y = map[triangles[i].y];
		triangles[i].z = map[triangles[i].z];
	}
}

__global__
void scatter_kernel(const int32_t *__restrict__ map, const glm::vec3 *__restrict__ data, glm::vec3 *__restrict__ dst, size_t size)
{
	size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
	size_t stride = blockDim.x * gridDim.x;

	for (size_t i = tid; i < size; i += stride) {
		int32_t index = map[i];
		dst[i] = data[index];
	}
}

struct remapper : std::unordered_map <int32_t, int32_t> {
	// CUDA map
	int32_t *dev_map = nullptr; // index -> value

	explicit remapper(const std::unordered_map <int32_t, int32_t> &map)
			: std::unordered_map <int32_t, int32_t> (map) {
		// Make sure that all values are present
		// i.e. from 1 to map size
		for (int32_t i = 0; i < map.size(); i++)
			assert(this->find(i) != this->end());

		// Allocate a device map
		std::vector <int32_t> host_map(map.size());
		for (auto &kv : map)
			host_map[kv.first] = kv.second;

		hipMalloc(&dev_map, map.size() * sizeof(int32_t));
		hipMemcpy(dev_map, host_map.data(), map.size() * sizeof(int32_t), hipMemcpyHostToDevice);
	}

	torch::Tensor remap(const torch::Tensor &indices) const {
		assert(indices.dtype() == torch::kInt32);
		assert(indices.is_cpu());

		torch::Tensor out = torch::zeros_like(indices);
		int32_t *out_ptr = out.data_ptr <int32_t> ();
		int32_t *indices_ptr = indices.data_ptr <int32_t> ();

		for (int32_t i = 0; i < indices.numel(); i++) {
			auto it = this->find(indices_ptr[i]);
			assert(it != this->end());
			out_ptr[i] = it->second;
		}

		return out;
	}

	torch::Tensor remap_device(const torch::Tensor &indices) const {
		assert(indices.dtype() == torch::kInt32);
		assert(indices.dim() == 2 && indices.size(1) == 3);
		assert(indices.is_cuda());

		torch::Tensor out = indices.clone();
		glm::ivec3 *out_ptr = (glm::ivec3 *) out.data_ptr <int32_t> ();

		dim3 block(256);
		dim3 grid((indices.size(0) + block.x - 1) / block.x);

		remapper_kernel <<< grid, block >>> (dev_map, out_ptr, indices.size(0));

		hipDeviceSynchronize();
		hipError_t err = hipGetLastError();
		if (err != hipSuccess) {
			std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
			exit(1);
		}

		return out;
	}

	torch::Tensor scatter(const torch::Tensor &vertices) const {
		assert(vertices.dtype() == torch::kFloat32);
		assert(vertices.dim() == 2 && vertices.size(1) == 3);
		assert(vertices.is_cpu());

		torch::Tensor out = torch::zeros_like(vertices);
		glm::vec3 *out_ptr = (glm::vec3 *) out.data_ptr <float> ();
		glm::vec3 *vertices_ptr = (glm::vec3 *) vertices.data_ptr <float> ();

		for (int32_t i = 0; i < vertices.size(0); i++) {
			auto it = this->find(i);
			assert(it != this->end());
			out_ptr[i] = vertices_ptr[it->second];
		}

		return out;
	}

	torch::Tensor scatter_device(const torch::Tensor &vertices) const {
		assert(vertices.dtype() == torch::kFloat32);
		assert(vertices.dim() == 2 && vertices.size(1) == 3);
		assert(vertices.is_cuda());

		torch::Tensor out = torch::zeros_like(vertices);
		glm::vec3 *out_ptr = (glm::vec3 *) out.data_ptr <float> ();
		glm::vec3 *vertices_ptr = (glm::vec3 *) vertices.data_ptr <float> ();

		dim3 block(256);
		dim3 grid((vertices.size(0) + block.x - 1) / block.x);

		scatter_kernel <<< grid, block >>> (dev_map, vertices_ptr, out_ptr, vertices.size(0));

		hipDeviceSynchronize();
		hipError_t err = hipGetLastError();
		if (err != hipSuccess) {
			std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
			exit(1);
		}

		return out;
	}
};

remapper generate_remapper(const torch::Tensor &complexes,
		std::unordered_map <int32_t, std::set <int32_t>> &cmap,
		int64_t vertex_count,
		int64_t sample_rate)
{
	assert(complexes.is_cpu());
	assert(complexes.dtype() == torch::kInt32);
	assert(complexes.dim() == 2 && complexes.size(1) == 4);

	std::vector <glm::ivec4> cs(complexes.size(0));
	// printf("cs: %lu\n", cs.size());
	int32_t *ptr = complexes.data_ptr <int32_t> ();
	std::memcpy(cs.data(), ptr, complexes.size(0) * sizeof(glm::ivec4));

	// Mappings
	std::unordered_map <int32_t, int32_t> rcmap;
	for (const auto &[k, v] : cmap) {
		for (const auto &i : v)
			rcmap[i] = k;
	}

	std::unordered_map <int32_t, int32_t> remap;
	// remapper remap;
	for (size_t i = 0; i < vertex_count; i++)
		remap[i] = i;

	for (const auto &[_, s] : cmap) {
		int32_t new_vertex = *s.begin();
		for (const auto &v : s)
			remap[v] = new_vertex;
	}

	std::unordered_map <ordered_pair, std::set <std::pair <int32_t, std::vector <int32_t>>>, ordered_pair::hash> bmap;

	for (int32_t i = 0; i < cs.size(); i++) {
		int32_t i00 = i * sample_rate * sample_rate;
		int32_t i10 = i00 + (sample_rate - 1);
		int32_t i01 = i00 + (sample_rate - 1) * sample_rate;
		int32_t i11 = i00 + (sample_rate * sample_rate - 1);

		int32_t c00 = rcmap[i00];
		int32_t c10 = rcmap[i10];
		int32_t c01 = rcmap[i01];
		int32_t c11 = rcmap[i11];

		ordered_pair p;
		bool reversed;

		std::vector <int32_t> b00_10;
		std::vector <int32_t> b00_01;
		std::vector <int32_t> b10_11;
		std::vector <int32_t> b01_11;

		// 00 -> 10
		reversed = p.from(c00, c10);
		if (reversed) {
			for (int32_t i = sample_rate - 2; i >= 1; i--)
				b00_10.push_back(i + i00);
		} else {
			for (int32_t i = 1; i <= sample_rate - 2; i++)
				b00_10.push_back(i + i00);
		}

		bmap[p].insert({ i, b00_10 });

		// 00 -> 01
		reversed = p.from(c00, c01);
		if (reversed) {
			for (int32_t i = sample_rate * (sample_rate - 2); i >= sample_rate; i -= sample_rate)
				b00_01.push_back(i + i00);
		} else {
			for (int32_t i = sample_rate; i <= sample_rate * (sample_rate - 2); i += sample_rate)
				b00_01.push_back(i + i00);
		}

		bmap[p].insert({ i, b00_01 });

		// 10 -> 11
		reversed = p.from(c10, c11);
		if (reversed) {
			for (int32_t i = sample_rate - 2; i >= 1; i--)
				b10_11.push_back(i * sample_rate + sample_rate - 1 + i00);
		} else {
			for (int32_t i = 1; i <= sample_rate - 2; i++)
				b10_11.push_back(i * sample_rate + sample_rate - 1 + i00);
		}

		bmap[p].insert({ i, b10_11 });

		// 01 -> 11
		reversed = p.from(c01, c11);
		if (reversed) {
			for (int32_t i = sample_rate - 2; i >= 1; i--)
				b01_11.push_back((sample_rate - 1) * sample_rate + i + i00);
		} else {
			for (int32_t i = 1; i <= sample_rate - 2; i++)
				b01_11.push_back((sample_rate - 1) * sample_rate + i + i00);
		}

		bmap[p].insert({ i, b01_11 });
	}

	for (const auto &[p, bs] : bmap) {
		const auto &ref = *bs.begin();
		for (const auto &b : bs) {
			for (int32_t i = 0; i < b.second.size(); i++) {
				remap[b.second[i]] = ref.second[i];
			}
		}
	}

	return remapper(remap);
}
