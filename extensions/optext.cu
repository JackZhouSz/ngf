#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstring>
#include <map>
#include <set>
#include <stdio.h>
#include <unordered_set>
#include <vector>
#include <queue>

#include "common.hpp"

// Closest point caching acceleration structure and arguments
struct dev_cached_grid {
	glm::vec3 min;
	glm::vec3 max;
	glm::vec3 bin_size;

	glm::vec3 *vertices = nullptr;
	glm::uvec3 *triangles = nullptr;

	uint32_t *query_triangles = nullptr;
	uint32_t *index0 = nullptr;
	uint32_t *index1 = nullptr;

	uint32_t vertex_count;
	uint32_t triangle_count;
	uint32_t resolution;

	bool good() const {
		return vertices != nullptr
			&& triangles != nullptr
			&& query_triangles != nullptr
			&& index0 != nullptr
			&& index1 != nullptr;
	}
};

struct cached_grid {
	geometry ref;

	glm::vec3 min;
	glm::vec3 max;

	uint32_t resolution;
	glm::vec3 bin_size;

	using query_bin = std::vector <uint32_t>;
	std::vector <query_bin> overlapping_triangles;
	std::vector <query_bin> query_triangles;

	dev_cached_grid dev_cas;

	// Construct from mesh
	cached_grid(const geometry &, uint32_t);

	uint32_t to_index(const glm::ivec3 &bin) const;
	uint32_t to_index(const glm::vec3 &p) const;

	std::unordered_set <uint32_t> closest_triangles(const glm::vec3 &p) const;

	bool precache_query(const glm::vec3 &p);
	// float precache_query(const std::vector <glm::vec3> &points);

	float precache_query_vector(const torch::Tensor &);
	float precache_query_vector_device(const torch::Tensor &);

	// Returns closest point, barycentric coordinates, distance, and triangle index
	std::tuple <glm::vec3, glm::vec3, float, uint32_t> query(const glm::vec3 &p) const;

	void query_vector(const torch::Tensor &,
		torch::Tensor &,
		torch::Tensor &,
		torch::Tensor &,
		torch::Tensor &) const;

	void query_vector_device(const torch::Tensor &,
		torch::Tensor &,
		torch::Tensor &,
		torch::Tensor &,
		torch::Tensor &) const;

	void precache_device();
};

// Bounding box of mesh
static std::pair <glm::vec3, glm::vec3> bound(const geometry &g)
{
	glm::vec3 max = g.vertices[0];
	glm::vec3 min = g.vertices[0];
	for (const glm::vec3 &v : g.vertices) {
		max = glm::max(max, v);
		min = glm::min(min, v);
	}

	return { max, min };
}

// Closest point on triangle
__forceinline__ __host__ __device__
static void triangle_closest_point(const glm::vec3 &v0, const glm::vec3 &v1, const glm::vec3 &v2, const glm::vec3 &p, glm::vec3 *closest, glm::vec3 *bary, float *distance)
{
	glm::vec3 B = v0;
	glm::vec3 E1 = v1 - v0;
	glm::vec3 E2 = v2 - v0;
	glm::vec3 D = B - p;

	float a = glm::dot(E1, E1);
	float b = glm::dot(E1, E2);
	float c = glm::dot(E2, E2);
	float d = glm::dot(E1, D);
	float e = glm::dot(E2, D);
	float f = glm::dot(D, D);

	float det = a * c - b * b;
	float s = b * e - c * d;
	float t = b * d - a * e;

	if (s + t <= det) {
		if (s < 0.0f) {
			if (t < 0.0f) {
				if (d < 0.0f) {
					s = glm::clamp(-d / a, 0.0f, 1.0f);
					t = 0.0f;
				} else {
					s = 0.0f;
					t = glm::clamp(-e / c, 0.0f, 1.0f);
				}
			} else {
				s = 0.0f;
				t = glm::clamp(-e / c, 0.0f, 1.0f);
			}
		} else if (t < 0.0f) {
			s = glm::clamp(-d / a, 0.0f, 1.0f);
			t = 0.0f;
		} else {
			float invDet = 1.0f / det;
			s *= invDet;
			t *= invDet;
		}
	} else {
		if (s < 0.0f) {
			float tmp0 = b + d;
			float tmp1 = c + e;
			if (tmp1 > tmp0) {
				float numer = tmp1 - tmp0;
				float denom = a - 2 * b + c;
				s = glm::clamp(numer / denom, 0.0f, 1.0f);
				t = 1 - s;
			} else {
				t = glm::clamp(-e / c, 0.0f, 1.0f);
				s = 0.0f;
			}
		} else if (t < 0.0f) {
			if (a + d > b + e) {
				float numer = c + e - b - d;
				float denom = a - 2 * b + c;
				s = glm::clamp(numer / denom, 0.0f, 1.0f);
				t = 1 - s;
			} else {
				s = glm::clamp(-e / c, 0.0f, 1.0f);
				t = 0.0f;
			}
		} else {
			float numer = c + e - b - d;
			float denom = a - 2 * b + c;
			s = glm::clamp(numer / denom, 0.0f, 1.0f);
			t = 1.0f - s;
		}
	}

	*closest = B + s * E1 + t * E2;
	*bary = glm::vec3(1.0f - s - t, s, t);
	*distance = glm::length(*closest - p);
}

// Cached acceleration structure
cached_grid::cached_grid(const geometry &ref_, uint32_t resolution_)
		: ref(ref_), resolution(resolution_)
{
	uint32_t size = resolution * resolution * resolution;
	overlapping_triangles.resize(size);
	query_triangles.resize(size);

	// Put triangles into bins
	std::tie(max, min) = bound(ref);
	glm::vec3 extent = { max.x - min.x, max.y - min.y, max.z - min.z };
	bin_size = extent / (float) resolution;

	for (size_t i = 0; i < ref.triangles.size(); i++) {
		const glm::uvec3 &triangle = ref.triangles[i];

		// Triangle belongs to all bins it intersects
		glm::vec3 v0 = ref.vertices[triangle.x];
		glm::vec3 v1 = ref.vertices[triangle.y];
		glm::vec3 v2 = ref.vertices[triangle.z];

		glm::vec3 tri_min = glm::min(glm::min(v0, v1), v2);
		glm::vec3 tri_max = glm::max(glm::max(v0, v1), v2);

		glm::vec3 min_bin = glm::clamp((tri_min - min) / bin_size, glm::vec3(0), glm::vec3(resolution - 1));
		glm::vec3 max_bin = glm::clamp((tri_max - min) / bin_size, glm::vec3(0), glm::vec3(resolution - 1));

		for (int x = min_bin.x; x <= max_bin.x; x++) {
			for (int y = min_bin.y; y <= max_bin.y; y++) {
				for (int z = min_bin.z; z <= max_bin.z; z++) {
					int index = x + y * resolution + z * resolution * resolution;
					overlapping_triangles[index].push_back(i);
				}
			}
		}
	}
}

uint32_t cached_grid::to_index(const glm::ivec3 &bin) const
{
	return bin.x + bin.y * resolution + bin.z * resolution * resolution;
}

uint32_t cached_grid::to_index(const glm::vec3 &p) const
{
	glm::vec3 bin_flt = glm::clamp((p - min) / bin_size, glm::vec3(0), glm::vec3(resolution - 1));
	glm::ivec3 bin = glm::ivec3(bin_flt);
	return to_index(bin);
}

// Find the complete set of query triangles for a point
std::unordered_set <uint32_t> cached_grid::closest_triangles(const glm::vec3 &p) const
{
	// Get the current bin
	glm::vec3 bin_flt = glm::clamp((p - min) / bin_size, glm::vec3(0), glm::vec3(resolution - 1));
	glm::ivec3 bin = glm::ivec3(bin_flt);
	uint32_t bin_index = to_index(p);

	// Find the closest non-empty bins
	std::vector <glm::ivec3> closest_bins;

	if (!overlapping_triangles[bin_index].empty()) {
		closest_bins.push_back(bin);
	} else {
		std::vector <glm::ivec3> plausible_bins;
		std::queue <glm::ivec3> queue;

		std::unordered_set <glm::ivec3> visited;
		bool stop = false;

		queue.push(bin);
		while (!queue.empty()) {
			glm::ivec3 current = queue.front();
			queue.pop();

			// If visited, continue
			if (visited.find(current) != visited.end())
				continue;

			visited.insert(current);

			// If non-empty, add to plausible bins and continue
			uint32_t current_index = current.x + current.y * resolution + current.z * resolution * resolution;
			if (!overlapping_triangles[current_index].empty()) {
				plausible_bins.push_back(current);

				// Also set the stop flag to stop adding neighbors
				stop = true;
				continue;
			}

			if (stop)
				continue;

			int dx[] = { -1, 0, 0, 1, 0, 0 };
			int dy[] = { 0, -1, 0, 0, 1, 0 };
			int dz[] = { 0, 0, -1, 0, 0, 1 };

			// Add all neighbors to queue...
			for (int i = 0; i < 6; i++) {
				glm::ivec3 next = current + glm::ivec3(dx[i], dy[i], dz[i]);
				if (next.x < 0 || next.x >= resolution ||
					next.y < 0 || next.y >= resolution ||
					next.z < 0 || next.z >= resolution)
					continue;

				// ...if not visited
				if (visited.find(next) == visited.end())
					queue.push(next);
			}
		}

		// Sort plausible bins by distance
		std::sort(plausible_bins.begin(), plausible_bins.end(),
			[&](const glm::ivec3 &a, const glm::ivec3 &b) {
				return glm::distance(bin_flt, glm::vec3(a)) < glm::distance(bin_flt, glm::vec3(b));
			}
		);

		assert(!plausible_bins.empty());

		// Add first one always; stop adding when difference is larger than voxel size
		closest_bins.push_back(plausible_bins[0]);
		for (uint32_t i = 1; i < plausible_bins.size(); i++) {
			glm::vec3 a = glm::vec3(plausible_bins[i - 1]);
			glm::vec3 b = glm::vec3(plausible_bins[i]);

			if (glm::distance(a, b) > 1.1f)
				break;

			closest_bins.push_back(plausible_bins[i]);
		}
	}

	assert(!closest_bins.empty());

	// Within the final collection, make sure to search immediate neighbors
	std::unordered_set <uint32_t> final_bins;

	for (const glm::ivec3 &bin : closest_bins) {
		int dx[] = { 0, -1, 0, 0, 1, 0, 0 };
		int dy[] = { 0, 0, -1, 0, 0, 1, 0 };
		int dz[] = { 0, 0, 0, -1, 0, 0, 1 };

		for (int i = 0; i < 7; i++) {
			glm::ivec3 next = bin + glm::ivec3(dx[i], dy[i], dz[i]);
			if (next.x < 0 || next.x >= resolution ||
				next.y < 0 || next.y >= resolution ||
				next.z < 0 || next.z >= resolution)
				continue;

			uint32_t next_index = to_index(next);
			if (!overlapping_triangles[next_index].empty())
				final_bins.insert(next_index);
		}
	}

	std::unordered_set <uint32_t> final_triangles;
	for (uint32_t bin_index : final_bins) {
		for (uint32_t index : overlapping_triangles[bin_index])
			final_triangles.insert(index);
	}

	return final_triangles;
}

// Load the cached query triangles if not already loaded
bool cached_grid::precache_query(const glm::vec3 &p)
{
	// Check if the bin is already cached
	uint32_t bin_index = to_index(p);
	if (!query_triangles[bin_index].empty())
		return false;

	// Otherwise, load the bin
	auto set = closest_triangles(p);
	query_triangles[bin_index] = query_bin(set.begin(), set.end());
	return true;
}

float cached_grid::precache_query_vector(const torch::Tensor &sources)
{
	// Ensure device and type and size
	assert(sources.dim() == 2 && sources.size(1) == 3);
	assert(sources.device().is_cpu());
	assert(sources.dtype() == torch::kFloat32);

	size_t size = sources.size(0);
	size_t any_count = 0;

	glm::vec3 *sources_ptr = (glm::vec3 *) sources.data_ptr <float> ();

	// #pragma omp parallel for reduction(+:any_count)
	for (uint32_t i = 0; i < size; i++) {
		any_count += precache_query(sources_ptr[i]);
	}

	return (float) any_count / (float) size;
}

float cached_grid::precache_query_vector_device(const torch::Tensor &sources)
{
	// Ensure device and type and size
	assert(sources.dim() == 2 && sources.size(1) == 3);
	assert(sources.device().is_cuda());
	assert(sources.dtype() == torch::kFloat32);

	size_t size = sources.size(0);
	size_t any_count = 0;

	glm::vec3 *sources_ptr_device = (glm::vec3 *) sources.data_ptr <float> ();
	glm::vec3 *sources_ptr = new glm::vec3[size];
	hipMemcpy(sources_ptr, sources_ptr_device, size * sizeof(glm::vec3), hipMemcpyDeviceToHost);

	// TODO: cuda check
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		std::cerr << "(precache) CUDA error: " << hipGetErrorString(err) << std::endl;
		exit(1);
	}

	// #pragma omp parallel for reduction(+:any_count)
	for (uint32_t i = 0; i < size; i++) {
		any_count += precache_query(sources_ptr[i]);
	}

	delete[] sources_ptr;
	return (float) any_count / (float) size;
}

// Single point query
std::tuple <glm::vec3, glm::vec3, float, uint32_t> cached_grid::query(const glm::vec3 &p) const
{
	// Assuming the point is precached already
	uint32_t bin_index = to_index(p);
	assert(bin_index < overlapping_triangles.size());

	const std::vector <uint32_t> &bin = query_triangles[bin_index];
	assert(bin.size() > 0);

	glm::vec3 closest = p;
	glm::vec3 barycentric;
	float distance = FLT_MAX;
	uint32_t triangle_index = 0;

	for (uint32_t index : bin) {
		const glm::uvec3 &tri = ref.triangles[index];
		glm::vec3 a = ref.vertices[tri[0]];
		glm::vec3 b = ref.vertices[tri[1]];
		glm::vec3 c = ref.vertices[tri[2]];

		glm::vec3 point;
		glm::vec3 bary;
		float dist;
		triangle_closest_point(a, b, c, p, &point, &bary, &dist);

		if (dist < distance) {
			closest = point;
			barycentric = bary;
			distance = dist;
			triangle_index = index;
		}
	}

	return std::make_tuple(closest, barycentric, distance, triangle_index);
}

void cached_grid::query_vector(const torch::Tensor &sources,
		torch::Tensor &closest,
		torch::Tensor &bary,
		torch::Tensor &distance,
		torch::Tensor &triangle_index) const
{
	// Check types, devices and sizes
	assert(sources.dim() == 2 && sources.size(1) == 3);
	assert(closest.dim() == 2 && closest.size(1) == 3);
	assert(bary.dim() == 2 && bary.size(1) == 3);
	assert(distance.dim() == 1);
	assert(triangle_index.dim() == 1);

	assert(sources.device().is_cpu());
	assert(closest.device().is_cpu());
	assert(bary.device().is_cpu());
	assert(distance.device().is_cpu());
	assert(triangle_index.device().is_cpu());

	assert(sources.dtype() == torch::kFloat32);
	assert(closest.dtype() == torch::kFloat32);
	assert(bary.dtype() == torch::kFloat32);
	assert(distance.dtype() == torch::kFloat32);
	assert(triangle_index.dtype() == torch::kInt32);

	assert(sources.size(0) == closest.size(0));
	assert(sources.size(0) == bary.size(0));
	assert(sources.size(0) == distance.size(0));
	assert(sources.size(0) == triangle_index.size(0));

	// Assuming all elements are precached already
	// and that the dst vector is already allocated
	size_t size = sources.size(0);

	glm::vec3 *sources_ptr = (glm::vec3 *) sources.data_ptr <float> ();
	glm::vec3 *closest_ptr = (glm::vec3 *) closest.data_ptr <float> ();
	glm::vec3 *bary_ptr = (glm::vec3 *) bary.data_ptr <float> ();
	float *distance_ptr = distance.data_ptr <float> ();
	int32_t *triangle_index_ptr = triangle_index.data_ptr <int32_t> ();

	#pragma omp parallel for
	for (uint32_t i = 0; i < size; i++) {
		uint32_t bin_index = to_index(sources_ptr[i]);
		auto [c, b, d, t] = query(sources_ptr[i]);

		closest_ptr[i] = c;
		bary_ptr[i] = b;
		distance_ptr[i] = d;
		triangle_index_ptr[i] = t;
	}
}

struct closest_point_kinfo {
	const glm::vec3 *__restrict__ points;
	glm::vec3       *__restrict__ closest;
	glm::vec3       *__restrict__ bary;
	float           *__restrict__ distances;
	int32_t         *__restrict__ triangles;

	int32_t                       point_count;
};

__global__
static void closest_point_kernel(dev_cached_grid cas, closest_point_kinfo kinfo)
{
	uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
	uint32_t stride = blockDim.x * gridDim.x;

	for (uint32_t i = tid; i < kinfo.point_count; i += stride) {
		glm::vec3 point = kinfo.points[i];
		glm::vec3 closest;
		uint32_t triangle;

		glm::vec3 bin_flt = glm::clamp((point - cas.min) / cas.bin_size,
				glm::vec3(0), glm::vec3(cas.resolution - 1));

		glm::ivec3 bin = glm::ivec3(bin_flt);
		uint32_t bin_index = bin.x + bin.y * cas.resolution + bin.z * cas.resolution * cas.resolution;

		uint32_t index0 = cas.index0[bin_index];
		uint32_t index1 = cas.index1[bin_index];

		glm::vec3 min_bary;
		float min_distance = FLT_MAX;

		for (uint32_t j = index0; j < index1; j++) {
			uint32_t triangle_index = cas.query_triangles[j];
			glm::uvec3 tri = cas.triangles[triangle_index];

			glm::vec3 v0 = cas.vertices[tri.x];
			glm::vec3 v1 = cas.vertices[tri.y];
			glm::vec3 v2 = cas.vertices[tri.z];

			// TODO: prune triangles that are too far away (based on bbox)?
			glm::vec3 candidate;
			glm::vec3 bary;
			float distance;

			triangle_closest_point(v0, v1, v2, point, &candidate, &bary, &distance);

			if (distance < min_distance) {
				closest = candidate;
				min_bary = bary;
				min_distance = distance;
				triangle = triangle_index;
			}
		}

		// TODO: barycentrics as well...
		kinfo.bary[i] = min_bary;
		kinfo.closest[i] = closest;
		kinfo.distances[i] = min_distance;
		kinfo.triangles[i] = triangle;
	}
}

void cached_grid::query_vector_device(const torch::Tensor &sources,
		torch::Tensor &closest,
		torch::Tensor &bary,
		torch::Tensor &distance,
		torch::Tensor &triangle_index) const
{
	// Check types, devices and sizes
	assert(sources.dim() == 2 && sources.size(1) == 3);
	assert(closest.dim() == 2 && closest.size(1) == 3);
	assert(bary.dim() == 2 && bary.size(1) == 3);
	assert(distance.dim() == 1);
	assert(triangle_index.dim() == 1);

	assert(sources.device().is_cuda());
	assert(closest.device().is_cuda());
	assert(bary.device().is_cuda());
	assert(distance.device().is_cuda());
	assert(triangle_index.device().is_cuda());

	assert(sources.dtype() == torch::kFloat32);
	assert(closest.dtype() == torch::kFloat32);
	assert(bary.dtype() == torch::kFloat32);
	assert(distance.dtype() == torch::kFloat32);
	assert(triangle_index.dtype() == torch::kInt32);

	assert(sources.size(0) == closest.size(0));
	assert(sources.size(0) == bary.size(0));
	assert(sources.size(0) == distance.size(0));
	assert(sources.size(0) == triangle_index.size(0));

	// Make sure the dev cas is up to date
	if (!dev_cas.good())
		throw std::runtime_error("(query_device) dev_cas is not good");

	// Assuming all elements are precached already (in device as well)
	// and that the dst vector is already allocated
	size_t size = sources.size(0);

	closest_point_kinfo kinfo;
	kinfo.points = (glm::vec3 *) sources.data_ptr <float> ();
	kinfo.closest = (glm::vec3 *) closest.data_ptr <float> ();
	kinfo.bary = (glm::vec3 *) bary.data_ptr <float> ();
	kinfo.distances = distance.data_ptr <float> ();
	kinfo.triangles = triangle_index.data_ptr <int32_t> ();
	kinfo.point_count = size;

	dim3 block(256);
	dim3 grid((size + block.x - 1) / block.x);

	closest_point_kernel <<< grid, block >>> (dev_cas, kinfo);
	hipDeviceSynchronize();
	// TODO: cuda check
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
		throw std::runtime_error("(query_device) CUDA error: " + std::string(hipGetErrorString(err)));
}

void cached_grid::precache_device()
{
	dev_cas.min = min;
	dev_cas.max = max;
	dev_cas.bin_size = bin_size;

	dev_cas.resolution = resolution;
	dev_cas.vertex_count = ref.vertices.size();
	dev_cas.triangle_count = ref.triangles.size();

	std::vector <uint32_t> linear_query_triangles;
	std::vector <uint32_t> index0;
	std::vector <uint32_t> index1;

	uint32_t size = resolution * resolution * resolution;
	uint32_t offset = 0;

	for (uint32_t i = 0; i < size; i++) {
		uint32_t query_size = query_triangles[i].size();
		linear_query_triangles.insert(linear_query_triangles.end(),
				query_triangles[i].begin(),
				query_triangles[i].end());

		index0.push_back(offset);
		index1.push_back(offset + query_size);
		offset += query_size;
	}

	// Free old memory
	if (dev_cas.vertices != nullptr)
		hipFree(dev_cas.vertices);

	if (dev_cas.triangles != nullptr)
		hipFree(dev_cas.triangles);

	if (dev_cas.query_triangles != nullptr)
		hipFree(dev_cas.query_triangles);

	if (dev_cas.index0 != nullptr)
		hipFree(dev_cas.index0);

	if (dev_cas.index1 != nullptr)
		hipFree(dev_cas.index1);

	// Allocate new memory
	// TODO: no need to keep reallocating
	hipMalloc(&dev_cas.vertices, sizeof(glm::vec3) * ref.vertices.size());
	hipMalloc(&dev_cas.triangles, sizeof(glm::uvec3) * ref.triangles.size());

	hipMalloc(&dev_cas.query_triangles, sizeof(uint32_t) * linear_query_triangles.size());
	hipMalloc(&dev_cas.index0, sizeof(uint32_t) * index0.size());
	hipMalloc(&dev_cas.index1, sizeof(uint32_t) * index1.size());

	hipMemcpy(dev_cas.vertices, ref.vertices.data(), sizeof(glm::vec3) * ref.vertices.size(), hipMemcpyHostToDevice);
	hipMemcpy(dev_cas.triangles, ref.triangles.data(), sizeof(glm::uvec3) * ref.triangles.size(), hipMemcpyHostToDevice);

	hipMemcpy(dev_cas.query_triangles, linear_query_triangles.data(), sizeof(uint32_t) * linear_query_triangles.size(), hipMemcpyHostToDevice);
	hipMemcpy(dev_cas.index0, index0.data(), sizeof(uint32_t) * index0.size(), hipMemcpyHostToDevice);
	hipMemcpy(dev_cas.index1, index1.data(), sizeof(uint32_t) * index1.size(), hipMemcpyHostToDevice);
}

struct cumesh {
	const glm::vec3 *vertices;
	const glm::uvec3 *triangles;

	uint32_t vertex_count = 0;
	uint32_t triangle_count = 0;
};

// TODO: const restrict...
__global__
static void barycentric_closest_point_kernel(cumesh cm, closest_point_kinfo kinfo)
{
	uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
	uint32_t stride = blockDim.x * gridDim.x;

	for (uint32_t i = tid; i < kinfo.point_count; i += stride) {
		glm::vec3 point = kinfo.points[i];
		glm::vec3 closest;
		glm::vec3 barycentrics;
		uint32_t triangle;

		float min_distance = FLT_MAX;
		for (uint32_t j = 0; j < cm.triangle_count; j++) {
			glm::uvec3 tri = cm.triangles[j];

			glm::vec3 v0 = cm.vertices[tri.x];
			glm::vec3 v1 = cm.vertices[tri.y];
			glm::vec3 v2 = cm.vertices[tri.z];

			glm::vec3 candidate;
			glm::vec3 bary;
			float distance;

			triangle_closest_point(v0, v1, v2, point, &candidate, &bary, &distance);

			if (distance < min_distance) {
				min_distance = distance;
				closest = candidate;
				barycentrics = bary;
				triangle = j;
			}
		}

		kinfo.bary[i] = barycentrics;
		kinfo.triangles[i] = triangle;
	}
}

void barycentric_closest_points(const torch::Tensor &vertices, const torch::Tensor &triangles, const torch::Tensor &sources, torch::Tensor &bary, torch::Tensor &indices)
{
	// Check types, devices and sizes
	assert(vertices.dim() == 2 && vertices.size(1) == 3);
	assert(triangles.dim() == 2 && triangles.size(1) == 3);
	assert(sources.dim() == 2 && sources.size(1) == 3);
	assert(bary.dim() == 2 && bary.size(1) == 3);
	assert(indices.dim() == 1);

	assert(vertices.device().is_cuda());
	assert(triangles.device().is_cuda());
	assert(sources.device().is_cuda());
	assert(bary.device().is_cuda());
	assert(indices.device().is_cuda());

	assert(vertices.dtype() == torch::kFloat32);
	assert(triangles.dtype() == torch::kInt32);
	assert(sources.dtype() == torch::kFloat32);
	assert(bary.dtype() == torch::kFloat32);
	assert(indices.dtype() == torch::kInt32);

	assert(sources.size(0) == bary.size(0));
	assert(sources.size(0) == indices.size(0));

	// Assuming all elements are precached already (in device as well)
	// and that the dst vector is already allocated
	size_t size = sources.size(0);

	cumesh cm;

	cm.vertices = (glm::vec3 *) vertices.data_ptr <float> ();
	cm.triangles = (glm::uvec3 *) triangles.data_ptr <int32_t> ();
	cm.vertex_count = vertices.size(0);
	cm.triangle_count = triangles.size(0);

	closest_point_kinfo kinfo;

	kinfo.points = (glm::vec3 *) sources.data_ptr <float> ();
	kinfo.bary = (glm::vec3 *) bary.data_ptr <float> ();
	kinfo.triangles = (int32_t *) indices.data_ptr <int32_t> ();
	kinfo.point_count = size;

	dim3 block(256);
	dim3 grid((size + block.x - 1) / block.x);

	barycentric_closest_point_kernel <<<grid, block>>> (cm, kinfo);

	// TODO: cuda check
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
		throw std::runtime_error("barycentric_closest_points: " + std::string(hipGetErrorString(err)));
}

__global__
void laplacian_smooth_kernel(glm::vec3 *result, glm::vec3 *vertices, int32_t *graph, uint32_t count, uint32_t max_adj, float factor)
{
	int32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid >= count)
		return;

	glm::vec3 sum = glm::vec3(0.0f);
	int32_t adj_count = graph[tid * max_adj];
	int32_t *adj = graph + tid * max_adj + 1;
	for (int32_t i = 0; i < adj_count; i++)
		sum += vertices[adj[i]];
	sum /= (float) adj_count;
	if (adj_count == 0)
		sum = vertices[tid];

	result[tid] = vertices[tid] + (sum - vertices[tid]) * factor;
}

struct vertex_graph {
	std::unordered_map <int32_t, std::unordered_set <int32_t>> graph;

	int32_t *dev_graph = nullptr;
		
	int32_t max = 0;
	int32_t max_adj = 0;

	void allocate_device_graph() {
		max = 0;
		max_adj = 0;

		for (auto &kv : graph) {
			max_adj = std::max(max_adj, (int32_t) kv.second.size());
			max = std::max(max, kv.first);
		}

		// Allocate a device graph
		int32_t graph_size = max * (max_adj + 1);
		hipMalloc(&dev_graph, graph_size * sizeof(int32_t));

		std::vector <uint32_t> host_graph(graph_size, 0);
		for (auto &kv : graph) {
			int32_t i = kv.first;
			int32_t j = 0;
			assert(i * max_adj + j < graph_size);
			host_graph[i * max_adj + j++] = kv.second.size();
			for (auto &adj : kv.second) {
				assert(i * max_adj + j < graph_size);
				host_graph[i * max_adj + j++] = adj;
			}
		}

		hipMemcpy(dev_graph, host_graph.data(), graph_size * sizeof(int32_t), hipMemcpyHostToDevice);
	}

	vertex_graph(const torch::Tensor &primitives) {
		assert(primitives.dim() == 2);
		assert(primitives.dtype() == torch::kInt32);
		assert(primitives.device().is_cpu());

		// if (primitives.size(1) == 3)
		// 	initialize_from_triangles(primitives);
		// else if (primitives.size(1) == 4)
		// 	initialize_from_quadrilaterals(primitives);
		if (primitives.size(1) == 4)
			initialize_from_quadrilaterals(primitives);
		else
			assert(false);
	}

	void initialize_from_triangles(const torch::Tensor &triangles) {
		assert(triangles.dim() == 2 && triangles.size(1) == 3);
		assert(triangles.dtype() == torch::kInt32);
		assert(triangles.device().is_cpu());

		int32_t triangle_count = triangles.size(0);

		for (uint32_t i = 0; i < triangle_count; i++) {
			int32_t v0 = triangles[i][0].item().to <int32_t> ();
			int32_t v1 = triangles[i][1].item().to <int32_t> ();
			int32_t v2 = triangles[i][2].item().to <int32_t> ();

			graph[v0].insert(v1);
			graph[v0].insert(v2);

			graph[v1].insert(v0);
			graph[v1].insert(v2);

			graph[v2].insert(v0);
			graph[v2].insert(v1);
		}

		allocate_device_graph();
	}

	void initialize_from_quadrilaterals(const torch::Tensor &quads) {
		assert(quads.dim() == 2 && quads.size(1) == 4);
		assert(quads.dtype() == torch::kInt32);
		assert(quads.device().is_cpu());

		int32_t quad_count = quads.size(0);

		for (uint32_t i = 0; i < quad_count; i++) {
			int32_t v0 = quads[i][0].item().to <int32_t> ();
			int32_t v1 = quads[i][1].item().to <int32_t> ();
			int32_t v2 = quads[i][2].item().to <int32_t> ();
			int32_t v3 = quads[i][3].item().to <int32_t> ();

			graph[v0].insert(v1);
			graph[v0].insert(v3);

			graph[v1].insert(v0);
			graph[v1].insert(v2);

			graph[v2].insert(v1);
			graph[v2].insert(v3);

			graph[v3].insert(v0);
			graph[v3].insert(v2);
		}

		allocate_device_graph();
	}

	~vertex_graph() {
		if (dev_graph)
			hipFree(dev_graph);
	}

	torch::Tensor smooth(const torch::Tensor &vertices, float factor) const {
		assert(vertices.dim() == 2 && vertices.size(1) == 3);
		assert(vertices.dtype() == torch::kFloat32);
		assert(vertices.device().is_cpu());
		assert(max < vertices.size(0));

		torch::Tensor result = torch::zeros_like(vertices);

		glm::vec3 *v = (glm::vec3 *) vertices.data_ptr <float> ();
		glm::vec3 *r = (glm::vec3 *) result.data_ptr <float> ();

		for (uint32_t i = 0; i <= max; i++) {
			if (graph.find(i) == graph.end())
				continue;

			glm::vec3 sum = glm::vec3(0.0f);
			for (auto j : graph.at(i))
				sum += v[j];
			sum /= (float) graph.at(i).size();

			r[i] = (1.0f - factor) * v[i] + factor * sum;
		}

		return result;
	}

	torch::Tensor smooth_device(const torch::Tensor &vertices, float factor) const {
		assert(vertices.dim() == 2 && vertices.size(1) == 3);
		assert(vertices.dtype() == torch::kFloat32);
		assert(vertices.device().is_cuda());
		assert(max < vertices.size(0));

		torch::Tensor result = torch::zeros_like(vertices);

		glm::vec3 *v = (glm::vec3 *) vertices.data_ptr <float> ();
		glm::vec3 *r = (glm::vec3 *) result.data_ptr <float> ();

		dim3 block(256);
		dim3 grid((vertices.size(0) + block.x - 1) / block.x);

		laplacian_smooth_kernel <<<grid, block>>> (r, v, dev_graph, vertices.size(0), max_adj, factor);

		return result;
	}
};

// torch::Tensor conformal_graph(const torch::Tensor &quads)
// {
// 	// Requries quads
// 	assert(quads.dim() == 2 && quads.size(1) == 4);
// 	assert(quads.dtype() == torch::kInt32);
// 	assert(quads.device().is_cpu());
//
// 	// First build the adjacency graph and record quad sharing
// 	std::unordered_map <int32_t, std::unordered_set <int32_t>> graph;
// 	std::unordered_map <int32_t, std::vector <glm::ivec4>> shared;
//
// 	uint32_t quad_count = quads.size(0);
//
// 	// Fill the graphs
// 	for (uint32_t i = 0; i < quad_count; i++) {
// 		int32_t v0 = quads[i][0].item().to <int32_t> ();
// 		int32_t v1 = quads[i][1].item().to <int32_t> ();
// 		int32_t v2 = quads[i][2].item().to <int32_t> ();
// 		int32_t v3 = quads[i][3].item().to <int32_t> ();
//
// 		graph[v0].insert(v1);
// 		graph[v0].insert(v3);
//
// 		graph[v1].insert(v0);
// 		graph[v1].insert(v2);
//
// 		graph[v2].insert(v1);
// 		graph[v2].insert(v3);
//
// 		graph[v3].insert(v0);
// 		graph[v3].insert(v2);
//
// 		glm::ivec4 quad = glm::ivec4(v0, v1, v2, v3);
// 		shared[v0].push_back(quad);
// 		shared[v1].push_back(quad);
// 		shared[v2].push_back(quad);
// 		shared[v3].push_back(quad);
// 	}
//
// 	// Collect all crossings; only vertices with valence 4, with opposite vertices not sharing quads
// 	std::vector <glm::ivec4> crossings;
//
// 	for (auto &kv : graph) {
// 		std::unordered_set <int32_t> &adj = kv.second;
// 		if (adj.size() != 4)
// 			continue;
//
// 		// Find the opposite vertices
// 		auto it = adj.begin();
// 		int32_t a = *it++;
// 		int32_t b = *it++;
// 		int32_t c = *it++;
// 		int32_t d = *it++;
//
// 		// printf("a: %d, b: %d, c: %d, d: %d\n", a, b, c, d);
// 		// printf("shared quads:\n");
// 		// for (auto &quad : shared[kv.first])
// 		// 	printf("  > %d %d %d %d\n", quad[0], quad[1], quad[2], quad[3]);
//
// 		// Expect that a-b and c-d are opposite (non-sharing), swap if otherwise
// 		int32_t opp_a = b;
//
// 		auto in_shared = [&shared](int32_t a, int32_t b) {
// 			for (auto &quad : shared[a])
// 				if (quad[0] == b || quad[1] == b || quad[2] == b || quad[3] == b)
// 					return true;
// 			return false;
// 		};
//
// 		if (in_shared(a, opp_a))
// 			opp_a = c;
// 		if (in_shared(a, opp_a))
// 			opp_a = d;
//
// 		assert(!in_shared(a, opp_a));
//
// 		// Then find the other two opposite vertices
// 		int32_t other_a = b;
// 		if (other_a == a || other_a == opp_a)
// 			other_a = c;
// 		if (other_a == a || other_a == opp_a)
// 			other_a = d;
//
// 		int32_t other_opp_a = b;
// 		if (other_opp_a == opp_a || other_opp_a == a || other_opp_a == other_a)
// 			other_opp_a = c;
// 		if (other_opp_a == opp_a || other_opp_a == a || other_opp_a == other_a)
// 			other_opp_a = d;
//
// 		assert(!in_shared(other_a, other_opp_a));
// 		assert(other_a != a && other_a != opp_a);
//
// 		// Finally, add the crossing
// 		crossings.push_back(glm::ivec4(a, opp_a, other_a, other_opp_a));
// 	}
//
// 	// Construct tensor of crossings
// 	torch::Tensor result = torch::zeros({ (long) crossings.size(), 4 }, torch::kInt32);
//
// 	for (uint32_t i = 0; i < crossings.size(); i++) {
// 		result[i][0] = crossings[i][0];
// 		result[i][1] = crossings[i][1];
// 		result[i][2] = crossings[i][2];
// 		result[i][3] = crossings[i][3];
// 	}
//
// 	return result;
// }

__global__
void triangulate_shorted_kernel(const glm::vec3 *__restrict__ vertices, glm::ivec3 *__restrict__ triangles, size_t sample_rate)
{
	size_t i = blockIdx.x;
	size_t j = threadIdx.x;
	size_t k = threadIdx.y;

	size_t offset = i * sample_rate * sample_rate;

	size_t a = offset + j * sample_rate + k;
	size_t b = a + 1;
	size_t c = offset + (j + 1) * sample_rate + k;
	size_t d = c + 1;

	const glm::vec3 &va = vertices[a];
	const glm::vec3 &vb = vertices[b];
	const glm::vec3 &vc = vertices[c];
	const glm::vec3 &vd = vertices[d];

	float d0 = glm::distance(va, vd);
	float d1 = glm::distance(vb, vc);

	size_t toffset = 2 * i * (sample_rate - 1) * (sample_rate - 1);
	size_t tindex = toffset + 2 * (j * (sample_rate - 1) + k);
	if (d0 < d1) {
		triangles[tindex] = glm::ivec3(a, d, b);
		triangles[tindex + 1] = glm::ivec3(a, c, d);
	} else {
		triangles[tindex] = glm::ivec3(a, c, b);
		triangles[tindex + 1] = glm::ivec3(b, c, d);
	}
}

torch::Tensor triangulate_shorted(const torch::Tensor &vertices, size_t complex_count, size_t sample_rate)
{
	assert(vertices.dtype() == torch::kFloat32);
	assert(vertices.dim() == 2 && vertices.size(1) == 3);
	assert(vertices.is_cuda());

	long triangle_count = 2 * complex_count * (sample_rate - 1) * (sample_rate - 1);

	auto options = torch::TensorOptions()
		.dtype(torch::kInt32)
		.device(torch::kCUDA, 0);

	torch::Tensor out = torch::zeros({ triangle_count, 3 }, options);

	glm::vec3 *vertices_ptr = (glm::vec3 *) vertices.data_ptr <float> ();
	glm::ivec3 *out_ptr = (glm::ivec3 *) out.data_ptr <int32_t> ();

	dim3 block(sample_rate - 1, sample_rate - 1);
	dim3 grid(complex_count);

	triangulate_shorted_kernel <<< grid, block >>> (vertices_ptr, out_ptr, sample_rate);

	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
		exit(1);
	}

	return out;
}

__global__
void remapper_kernel(const int32_t *__restrict__ map, glm::ivec3 *__restrict__ triangles, size_t size)
{
	size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
	size_t stride = blockDim.x * gridDim.x;

	for (size_t i = tid; i < size; i += stride) {
		triangles[i].x = map[triangles[i].x];
		triangles[i].y = map[triangles[i].y];
		triangles[i].z = map[triangles[i].z];
	}
}

__global__
void scatter_kernel(const int32_t *__restrict__ map, const glm::vec3 *__restrict__ data, glm::vec3 *__restrict__ dst, size_t size)
{
	size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
	size_t stride = blockDim.x * gridDim.x;

	for (size_t i = tid; i < size; i += stride) {
		int32_t index = map[i];
		dst[i] = data[index];
	}
}

struct remapper : std::unordered_map <int32_t, int32_t> {
	// CUDA map
	int32_t *dev_map = nullptr; // index -> value

	explicit remapper(const std::unordered_map <int32_t, int32_t> &map)
			: std::unordered_map <int32_t, int32_t> (map) {
		// Make sure that all values are present
		// i.e. from 1 to map size
		for (int32_t i = 0; i < map.size(); i++)
			assert(this->find(i) != this->end());

		// Allocate a device map
		std::vector <int32_t> host_map(map.size());
		for (auto &kv : map)
			host_map[kv.first] = kv.second;

		hipMalloc(&dev_map, map.size() * sizeof(int32_t));
		hipMemcpy(dev_map, host_map.data(), map.size() * sizeof(int32_t), hipMemcpyHostToDevice);
	}

	torch::Tensor remap(const torch::Tensor &indices) const {
		assert(indices.dtype() == torch::kInt32);
		assert(indices.is_cpu());

		torch::Tensor out = torch::zeros_like(indices);
		int32_t *out_ptr = out.data_ptr <int32_t> ();
		int32_t *indices_ptr = indices.data_ptr <int32_t> ();

		for (int32_t i = 0; i < indices.numel(); i++) {
			auto it = this->find(indices_ptr[i]);
			assert(it != this->end());
			out_ptr[i] = it->second;
		}

		return out;
	}

	torch::Tensor remap_device(const torch::Tensor &indices) const {
		assert(indices.dtype() == torch::kInt32);
		assert(indices.dim() == 2 && indices.size(1) == 3);
		assert(indices.is_cuda());

		torch::Tensor out = indices.clone();
		glm::ivec3 *out_ptr = (glm::ivec3 *) out.data_ptr <int32_t> ();

		dim3 block(256);
		dim3 grid((indices.size(0) + block.x - 1) / block.x);

		remapper_kernel <<< grid, block >>> (dev_map, out_ptr, indices.size(0));

		hipDeviceSynchronize();
		hipError_t err = hipGetLastError();
		if (err != hipSuccess) {
			std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
			exit(1);
		}

		return out;
	}

	torch::Tensor scatter(const torch::Tensor &vertices) const {
		assert(vertices.dtype() == torch::kFloat32);
		assert(vertices.dim() == 2 && vertices.size(1) == 3);
		assert(vertices.is_cpu());

		torch::Tensor out = torch::zeros_like(vertices);
		glm::vec3 *out_ptr = (glm::vec3 *) out.data_ptr <float> ();
		glm::vec3 *vertices_ptr = (glm::vec3 *) vertices.data_ptr <float> ();

		for (int32_t i = 0; i < vertices.size(0); i++) {
			auto it = this->find(i);
			assert(it != this->end());
			out_ptr[i] = vertices_ptr[it->second];
		}

		return out;
	}

	torch::Tensor scatter_device(const torch::Tensor &vertices) const {
		assert(vertices.dtype() == torch::kFloat32);
		assert(vertices.dim() == 2 && vertices.size(1) == 3);
		assert(vertices.is_cuda());

		torch::Tensor out = torch::zeros_like(vertices);
		glm::vec3 *out_ptr = (glm::vec3 *) out.data_ptr <float> ();
		glm::vec3 *vertices_ptr = (glm::vec3 *) vertices.data_ptr <float> ();

		dim3 block(256);
		dim3 grid((vertices.size(0) + block.x - 1) / block.x);

		scatter_kernel <<< grid, block >>> (dev_map, vertices_ptr, out_ptr, vertices.size(0));

		hipDeviceSynchronize();
		hipError_t err = hipGetLastError();
		if (err != hipSuccess) {
			std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
			exit(1);
		}

		return out;
	}
};

remapper generate_remapper(const torch::Tensor &complexes,
		std::unordered_map <int32_t, std::set <int32_t>> &cmap,
		int64_t vertex_count,
		int64_t sample_rate)
{
	assert(complexes.is_cpu());
	assert(complexes.dtype() == torch::kInt32);
	assert(complexes.dim() == 2 && complexes.size(1) == 4);

	std::vector <glm::ivec4> cs(complexes.size(0));
	// printf("cs: %lu\n", cs.size());
	int32_t *ptr = complexes.data_ptr <int32_t> ();
	std::memcpy(cs.data(), ptr, complexes.size(0) * sizeof(glm::ivec4));

	// Mappings
	std::unordered_map <int32_t, int32_t> rcmap;
	for (const auto &[k, v] : cmap) {
		for (const auto &i : v)
			rcmap[i] = k;
	}

	std::unordered_map <int32_t, int32_t> remap;
	// remapper remap;
	for (size_t i = 0; i < vertex_count; i++)
		remap[i] = i;

	for (const auto &[_, s] : cmap) {
		int32_t new_vertex = *s.begin();
		for (const auto &v : s)
			remap[v] = new_vertex;
	}

	std::unordered_map <ordered_pair, std::set <std::pair <int32_t, std::vector <int32_t>>>, ordered_pair::hash> bmap;

	for (int32_t i = 0; i < cs.size(); i++) {
		int32_t i00 = i * sample_rate * sample_rate;
		int32_t i10 = i00 + (sample_rate - 1);
		int32_t i01 = i00 + (sample_rate - 1) * sample_rate;
		int32_t i11 = i00 + (sample_rate * sample_rate - 1);

		int32_t c00 = rcmap[i00];
		int32_t c10 = rcmap[i10];
		int32_t c01 = rcmap[i01];
		int32_t c11 = rcmap[i11];

		ordered_pair p;
		bool reversed;

		std::vector <int32_t> b00_10;
		std::vector <int32_t> b00_01;
		std::vector <int32_t> b10_11;
		std::vector <int32_t> b01_11;

		// 00 -> 10
		reversed = p.from(c00, c10);
		if (reversed) {
			for (int32_t i = sample_rate - 2; i >= 1; i--)
				b00_10.push_back(i + i00);
		} else {
			for (int32_t i = 1; i <= sample_rate - 2; i++)
				b00_10.push_back(i + i00);
		}

		bmap[p].insert({ i, b00_10 });

		// 00 -> 01
		reversed = p.from(c00, c01);
		if (reversed) {
			for (int32_t i = sample_rate * (sample_rate - 2); i >= sample_rate; i -= sample_rate)
				b00_01.push_back(i + i00);
		} else {
			for (int32_t i = sample_rate; i <= sample_rate * (sample_rate - 2); i += sample_rate)
				b00_01.push_back(i + i00);
		}

		bmap[p].insert({ i, b00_01 });

		// 10 -> 11
		reversed = p.from(c10, c11);
		if (reversed) {
			for (int32_t i = sample_rate - 2; i >= 1; i--)
				b10_11.push_back(i * sample_rate + sample_rate - 1 + i00);
		} else {
			for (int32_t i = 1; i <= sample_rate - 2; i++)
				b10_11.push_back(i * sample_rate + sample_rate - 1 + i00);
		}

		bmap[p].insert({ i, b10_11 });

		// 01 -> 11
		reversed = p.from(c01, c11);
		if (reversed) {
			for (int32_t i = sample_rate - 2; i >= 1; i--)
				b01_11.push_back((sample_rate - 1) * sample_rate + i + i00);
		} else {
			for (int32_t i = 1; i <= sample_rate - 2; i++)
				b01_11.push_back((sample_rate - 1) * sample_rate + i + i00);
		}

		bmap[p].insert({ i, b01_11 });
	}

	for (const auto &[p, bs] : bmap) {
		const auto &ref = *bs.begin();
		for (const auto &b : bs) {
			for (int32_t i = 0; i < b.second.size(); i++) {
				remap[b.second[i]] = ref.second[i];
			}
		}
	}

	return remapper(remap);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m)
{
        py::class_ <geometry> (m, "geometry")
                .def(py::init <const torch::Tensor &, const torch::Tensor &> ())
                .def(py::init <const torch::Tensor &, const torch::Tensor &, const torch::Tensor &> ())
		.def("deduplicate", &geometry::deduplicate)
		.def("torched", &geometry::torched)
		.def_readonly("vertices", &geometry::vertices)
		.def_readonly("normals", &geometry::normals)
		.def_readonly("triangles", &geometry::triangles)
		.def("__repr__", [](const geometry &g) {
			return "geometry(vertices=" + std::to_string(g.vertices.size())
				+ ", triangles=" + std::to_string(g.triangles.size()) + ")";
		});

	py::class_ <cached_grid> (m, "cached_grid")
		.def(py::init <const geometry &, uint32_t> ())
		.def("precache_query", &cached_grid::precache_query_vector)
		.def("precache_query_device", &cached_grid::precache_query_vector_device)
		.def("precache_device", &cached_grid::precache_device)
		.def("query", &cached_grid::query_vector)
		.def("query_device", &cached_grid::query_vector_device);

	py::class_ <vertex_graph> (m, "vertex_graph")
		.def(py::init <const torch::Tensor &> ())
		.def("smooth", &vertex_graph::smooth)
		.def("smooth_device", &vertex_graph::smooth_device);

	py::class_ <remapper> (m, "remapper")
		.def("remap", &remapper::remap, "Remap indices")
		.def("remap_device", &remapper::remap_device, "Remap indices")
		.def("scatter", &remapper::scatter, "Scatter vertex data")
		.def("scatter_device", &remapper::scatter_device, "Scatter vertex data");

	// m.def("conformal_graph", &conformal_graph);
	m.def("cluster_geometry", &cluster_geometry);
	m.def("barycentric_closest_points", &barycentric_closest_points);
	m.def("triangulate_shorted", &triangulate_shorted);
	m.def("generate_remapper", &generate_remapper, "Generate remapper");
}
